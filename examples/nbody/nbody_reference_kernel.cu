
#include <hip/hip_runtime.h>
inline __device__ float4 make_float4(float s)
{
    return make_float4(s, s, s, s);
}

inline __device__ float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w);
}

inline __host__ __device__ float4 operator-(float4 a, float4 b)
{
    return make_float4(a.x - b.x, a.y - b.y, a.z - b.z,  a.w - b.w);
}

inline __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    a.w += b.w;
}

inline __device__ float4 operator*(float4 a, float4 b)
{
    return make_float4(a.x * b.x, a.y * b.y, a.z * b.z,  a.w * b.w);
}

inline __device__ float4 operator*(float4 a, float b)
{
    return make_float4(a.x * b, a.y * b, a.z * b,  a.w * b);
}

inline __device__ float4 operator*(float b, float4 a)
{
    return make_float4(b * a.x, b * a.y, b * a.z, b * a.w);
}

extern "C" __global__ void nbody_kernel(float dt1,
	float4* pos_old, 
	float4* pos_new,
	float4* oldVel,
	float4* newVel,
	float damping, 
	float softeningSqr)
{
	const float4 dt = make_float4(dt1, dt1, dt1, 0.0f);//(float4){.x=dt1,.y=dt1,.z=dt1,.w=0.0f};
	int gti = blockIdx.x*blockDim.x + threadIdx.x;
	int ti = threadIdx.x;
	int n = blockDim.x*gridDim.x;
	int nt = blockDim.x;
	int nb = n/nt;
	__shared__ float4 pblock[1024]; // FIXME
	float4 p = pos_old[gti];
	float4 v = oldVel[gti];
	float4 a = make_float4(0.0f);//{.x=0.0f,.y=0.0f,.z=0.0f,.w=0.0f};
	
	for(int jb=0; jb < nb; jb++) { /* Foreach block ... */
		pblock[ti] = pos_old[jb*nt+ti]; /* Cache ONE particle position */
		__syncthreads(); /* Wait for others in the work-group */
		for(int j=0; j<nt; j++) { /* For ALL cached particle positions ... */
			float4 p2 = pblock[j]; /* Read a cached particle position */
			float4 d = p2 - p;
			float invr = rsqrtf(d.x*d.x + d.y*d.y + d.z*d.z + softeningSqr);
			float f = p2.w*invr*invr*invr;
			a += f*d; /* Accumulate acceleration */
		}
		__syncthreads(); /* Wait for others in work-group */
	}
	p += dt*v + damping*dt*dt*a;
	v += dt*a;

	pos_new[gti] = p;
	newVel[gti] = v;
}

